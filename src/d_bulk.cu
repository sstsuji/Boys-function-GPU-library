#include <cstdio>
#include <string>
#include <map>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include "mp.h"
#include "device.cuh"
#include "constants.h"


void deviceBulkEvaluation(Degree* h_n, double* h_x, double* h_boys, 
                          std::string scenario, const TaylorTable& h_LUT, 
                          int num_inputs, int num_evals)
{
    Degree* d_n; 
    double* d_x;
    double* d_x_sorted;
    double* d_boys_grid;
    Sortkey* d_key; 
    Sortkey* d_key_sorted;
    double* d_boys; 
    int* d_counter;

    const size_t n_size = sizeof(Degree) * num_inputs;
    const size_t x_size = sizeof(double) * num_inputs;
    size_t boys_size = 0;
    if (scenario == "single") {
        boys_size = sizeof(double) * num_inputs;
    }
    else if (scenario == "incremental") {
        boys_size = sizeof(double) * (h_LUT.n_max + 1) * num_inputs;
    }
    const size_t key_size = sizeof(Sortkey) * num_inputs;
    const size_t counter_size = sizeof(int);

    hipMalloc(&d_n, n_size);
    hipMalloc(&d_x, x_size);
    hipMalloc(&d_x_sorted, x_size);
    hipMalloc(&d_boys_grid, h_LUT.table_size);
    hipMalloc(&d_key, key_size);
    hipMalloc(&d_key_sorted, key_size);
    hipMalloc(&d_boys, boys_size);
    hipMalloc(&d_counter, counter_size);

    hipMemcpy(d_n, h_n, n_size, hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, x_size, hipMemcpyHostToDevice);
    hipMemcpy(d_boys_grid, h_LUT.boys_grid, h_LUT.table_size, hipMemcpyHostToDevice);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(&singleBulkEvaluation), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(&incrementalBulkEvaluation), hipFuncCachePreferL1);

    // allocate temporary memory for sorting
    void *d_tmp = NULL;
    size_t d_tmp_size = 0;
    hipcub::DeviceRadixSort::SortPairsDescending(d_tmp, d_tmp_size, 
                                              d_key, d_key_sorted, 
                                              d_x, d_x_sorted, num_inputs);
    hipMalloc(&d_tmp, d_tmp_size);

    const int num_samples = 10;
    std::map<std::string, float> kernel_time;
    kernel_time["key"] = 0.0f;
    kernel_time["sort"] = 0.0f;
    kernel_time["boys"] = 0.0f;

    const int threadsPerWarp = 32;
    const int threadsPerBlock = 128;
    const int warpsPerBlock = threadsPerBlock / threadsPerWarp;
    const int num_blocks = num_inputs / threadsPerBlock;
    dim3 blocks(num_blocks);
    dim3 threads(threadsPerWarp, warpsPerBlock);

    const int tpb = 1024;
    const int nb = num_inputs / tpb;

    hipEvent_t begin, end;
    float elapsed_time = 0.0f;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    for (int s = 0; s < num_samples; ++s) {
        hipMemset(d_boys, 0, boys_size);
        hipMemset(d_counter, 0, counter_size);

        //*
        if (scenario == "single") {
            hipEventRecord(begin);
            generateKeySingle<<<nb, tpb>>>(d_n, d_x, d_key);
            hipEventRecord(end);
        }
        else if (scenario == "incremental") {
            hipEventRecord(begin);
            generateKeyIncremental<<<nb, tpb>>>(d_n, d_x, d_key);
            hipEventRecord(end);
        }
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, begin, end);
        kernel_time["key"] += elapsed_time;
        /**/

        hipDeviceSynchronize();

        /*
        hipEventRecord(begin);
        hipcub::DeviceRadixSort::SortPairsDescending(d_tmp, d_tmp_size, 
                                                  d_key, d_key_sorted, 
                                                  d_x, d_x_sorted, num_inputs);
        hipEventRecord(end);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, begin, end);
        kernel_time["sort"] += elapsed_time;
        /**/

        //*
        if (scenario == "single") {
            hipEventRecord(begin);
            singleBulkEvaluation<<<blocks, threads>>>(d_key, d_x, d_boys, d_boys_grid, d_counter);
            // singleBulkEvaluation<<<blocks, threads>>>(d_key_sorted, d_x_sorted, d_boys, d_boys_grid, d_counter);
            hipEventRecord(end);
        }
        else if (scenario == "incremental") {
            hipEventRecord(begin);
            incrementalBulkEvaluation<<<blocks, threads>>>(d_key, d_x, d_boys, d_boys_grid, d_counter, num_inputs);
            // incrementalBulkEvaluation<<<blocks, threads>>>(d_key_sorted, d_x_sorted, d_boys, d_boys_grid, d_counter, num_inputs);
            hipEventRecord(end);
        }
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, begin, end);
        kernel_time["boys"] += elapsed_time;
        /**/
    }
    hipEventDestroy(begin);
    hipEventDestroy(end);
    hipDeviceSynchronize();

    hipMemcpy(h_boys, d_boys, boys_size, hipMemcpyDeviceToHost);

    kernel_time["key"] /= num_samples;
    kernel_time["sort"] /= num_samples;
    kernel_time["boys"] /= num_samples;
    printf("---<DEVICE COMPUTATION TIME>---\n");
    printf("key generation: %.2f [ms]\n", kernel_time["key"]);
    printf("input sorting: %.2f [ms]\n", kernel_time["sort"]);
    printf("boys evaluation: %.2f [ms]\n", kernel_time["boys"]);

    hipFree(d_n);
    hipFree(d_x);
    hipFree(d_x_sorted);
    hipFree(d_key);
    hipFree(d_key_sorted);
    hipFree(d_boys_grid);
    hipFree(d_boys);
    hipFree(d_counter);
    hipFree(d_tmp);
}


